#include "hip/hip_runtime.h"
/**
 * cuYASHE
 * Copyright (C) 2015-2016 cuYASHE Authors
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "cuda_distribution.h"

__global__ void setup_kernel ( hiprandState * states, unsigned long seed ){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init ( seed+tid*4, tid, 0, &states[tid] );
}

__host__ void Distribution::call_setup_kernel(){
	const int N = MAX_DEGREE;
	const int ADDGRIDXDIM = (N%ADDBLOCKXDIM == 0? N/ADDBLOCKXDIM : N/ADDBLOCKXDIM + 1);
	const dim3 gridDim(ADDGRIDXDIM);
	const dim3 blockDim(ADDBLOCKXDIM);

	setup_kernel<<<gridDim,blockDim,0>>>(states,SEED);
	assert(hipGetLastError() == hipSuccess);
}


__global__ void generate_narrow_random_numbers(	bn_t *coefs,
												hiprandState *states,
												int N,
												int spacing,
												int NPrimes,
												int mod ) {

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid <= N){	
    	int value = llrintf(hiprand_uniform(&states[tid])); // [-1, 0 , 1];
	value += (tid == N);
    	// value -= llrintf(hiprand_uniform(&states[tid]));
    	
    	// if(value == 0 && tid == N)
    	// 	value = 1;

    	// for(int i = 0; i < NPrimes; i++)
    		// coefs[tid + spacing*i] = value % CRTPrimesConstant[i];
    	coefs[tid].dp[0] = value;
    	coefs[tid].used = 1;
    	bn_zero_non_used(&coefs[tid]);
    }
        
}

__host__  void Distribution::callCuGetUniformSample(	bn_t *coefs,
														int N,
														int NPrimes,
														int mod ){
	/**
	 * Generates N random integers
	 */
	
	const int ADDGRIDXDIM = (N%ADDBLOCKXDIM == 0? N/ADDBLOCKXDIM : N/ADDBLOCKXDIM + 1);
	const dim3 gridDim(ADDGRIDXDIM);
	const dim3 blockDim(ADDBLOCKXDIM);

	/** 
	 * Generate values
	 */
	assert(N <= MAX_DEGREE);
	generate_narrow_random_numbers<<<gridDim,blockDim,0,NULL>>>( 	coefs,
																	states,
																	N,
																	CUDAFunctions::N,
																	NPrimes,
																	mod );
	assert(hipGetLastError() == hipSuccess);
}

__global__ void generate_normal_random_numbers(	bn_t *coefs,
												hiprandState *states,
												int N,
												int spacing,
												float mean, 
												float stddev,
												int NPrimes) {

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N){	
    	int value = hiprand_normal (&states[tid])*stddev + mean; 
	value += (tid == N);
    	// for(int i = 0; i < NPrimes; i++){
    		// coefs[tid + spacing*i] = value % CRTPrimesConstant[i];
    	// }
    	coefs[tid].dp[0] = value;
    	coefs[tid].used = 1;
    	bn_zero_non_used(&coefs[tid]);
    }
        
}

__host__ void Distribution::callCuGetNormalSample(	bn_t *coefs,
													int N,
													float mean,
													float stddev,
													int NPrimes){
		/**
	 * Generates N random integers
	 */
	
	const int ADDGRIDXDIM = (N%ADDBLOCKXDIM == 0? N/ADDBLOCKXDIM : N/ADDBLOCKXDIM + 1);
	const dim3 gridDim(ADDGRIDXDIM);
	const dim3 blockDim(ADDBLOCKXDIM);

	/** 
	 * Generate values
	 */
	assert(N <= MAX_DEGREE);
	generate_normal_random_numbers<<<blockDim,gridDim,0,NULL>>>( 	coefs,
																	states,
																	N,
																	CUDAFunctions::N,
																	mean,
																	stddev,
																	NPrimes );
	assert(hipGetLastError() == hipSuccess);
	


}
